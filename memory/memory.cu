/*
 * An exercise on the different types of memory available in CUDA
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// Error checking macro function
#define myCudaCheck(result) { cudaErrorCheck((result), __FILE__, __LINE__); }
inline void cudaErrorCheck(hipError_t err, const char* file, int line)
{
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << file << ":" << line << std::endl;
    exit(err);
  }
}

// Array size
// HANDSON 2.1 Change the array size to a static __constant__ int
//#define ARRAY_SIZE 65536
static __constant__ int arraysize; 

// CUDA threads per block
#define nThreads 128

// Array reversing kernel
__global__
void reverse(float* devA, float* devB)
{
  // HANDSON 2.3 Create a __shared__ temporary array of length nThreads for the swap
  __shared__ float temp[nThreads];

  // Get the index in this block
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // HANDSON 2.4 Fill the temporary array
  temp[nThreads - (threadIdx.x + 1)] = devA[idx];

  // HANDSON 2.5 synchronize the threads
  __syncthreads(); 

  // HANDSON 2.6 Calculate the initial position of this block in the grid
  int blockOffset = arraysize - (blockIdx.x + 1) * blockDim.x;

  // HANDSON 2.7 Fill the output array with the reversed elements from this block
  devB[blockOffset + threadIdx.x] = temp[threadIdx.x];
}

// Main host function
int main( )
{
  // HANDSON 2.2 Replace the host array size by a const int
  const int host_arraysize = 65536;
  // size of the array in char
  size_t sizeChar = host_arraysize * sizeof(float);

  // Allocate host memory
  float* hostIn = (float*) malloc(sizeChar);
  float* hostOut = (float*) malloc(sizeChar);

  // Allocate device memory
  float* devIn;
  float* devOut;
  myCudaCheck(
	      hipMalloc(&devIn, sizeChar)
	      );
  myCudaCheck(
	      hipMalloc(&devOut, sizeChar)
	      );

  // Initialize the arrays
  for (int i = 0; i < host_arraysize; i++) {
    hostIn[i] = i;
    hostOut[i] = 0;
  }

  // Copy the input array from the host to the device
  myCudaCheck(
	      hipMemcpy(devIn, hostIn, sizeChar, hipMemcpyHostToDevice)
	      );

  // Define the size of the task
  dim3 blocksPerGrid(host_arraysize/nThreads);
  dim3 threadsPerBlock(nThreads);

  reverse<<<blocksPerGrid, threadsPerBlock>>>(devIn, devOut);

  // Wait for all threads to complete
  myCudaCheck(
	      hipDeviceSynchronize()
	      );

  // Copy the result array back to the host
  myCudaCheck(
	      hipMemcpy(hostOut, devOut, sizeChar, hipMemcpyDeviceToHost)
	      );

  // Check and print the result
  int nCorrect = 0;
  for (int i = 0; i < host_arraysize; i++) {
    nCorrect += (hostOut[i] == hostIn[host_arraysize - (i+1)]) ? 1 : 0;
  }
  std::cout << ((nCorrect == host_arraysize) ? "Success! " : "Failure: ");
  std::cout << nCorrect  << " elements were correctly swapped." << std::endl;

  // Free device memory
  myCudaCheck(
	      hipFree(devIn)
	      );
  myCudaCheck(
	      hipFree(devOut)
	      );

  // Free host memory
  free(hostIn);
  free(hostOut);

  return 0;
}